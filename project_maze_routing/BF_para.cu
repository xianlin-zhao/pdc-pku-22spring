#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>


using namespace std;

#define N 10000
#define BLK -1
#define inf 1 << 30
#define FILENUM 5

// 分别是每个grid中的block数，每个block中的线程数。一共1e8个格子，大约是1<<27，每个block中设1<<9个线程，所以block数设置为1<<18
#define BLOCK_NUM 1 << 18
#define THREAD_NUM 512

// double consume_time[FILENUM]{};

/*
 * kernel函数
 * 前4个参数：迷宫矩阵、每个点距源点的距离、每个点的前驱、临时的dist（防止读写竞争，先把结果写到别处，最后再复制到dist里）
 * 后4个参数：本轮循环中是否有dist改变、四个方向的dx、四个方向的dy，矩阵边长（就是N，但是要搬运到cuda上）
 */
__global__ void BF_algo(short *d_maze, int *d_dist, int *d_prev, int *d_dist_tmp, 
                        bool *d_has_next, int *d_dx, int *d_dy, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;  // 线程的全局id

    if (tid >= n * n || d_maze[tid] == BLK)  // 是否超过了格子的最大编号？是否本身是障碍物？
        return;
    int i = tid / n, j = tid % n;  // 计算出二维坐标

    for (int k = 0; k < 4; k++) {  // 探测四个方向的边
        int newx = i + d_dx[k], newy = j + d_dy[k];
        if (newx < 0 || newx >= n || newy < 0 || newy >= n)
            continue;
        if (d_maze[newx * n + newy] == BLK)
            continue;
        int newdist = d_dist[newx * n + newy] + d_maze[i * n + j];
        if (newdist < d_dist[i * n + j]) {  // 松弛操作，新的距离比原来的dist小，更新
            d_dist_tmp[i * n + j] = newdist, d_prev[i * n + j] = newx * n + newy;
            *d_has_next = true;  // 有dist被更新了，所以还会有下一轮循环
        }
    }
}

int main()
{
    ofstream fout;
    fout.open("result.txt");
    short *maze = (short *) malloc(sizeof(short) * N * N);
    int *dist = (int *) malloc(sizeof(int) * N * N);
    int *prev = (int *) malloc(sizeof(int) * N * N);
    int *dist_tmp = (int *) malloc(sizeof(int) * N * N);  // dist的副本，所有线程结束后，把它复制到dist里

    for (int casenum = 0; casenum < FILENUM; casenum++) {
        ifstream fin;
        string str;
        // char num_str[10];
        // itoa(casenum, num_str, 10);
        // string case_path = "./benchmark/case" + string(num_str) + ".txt";
        string case_path = "./benchmark/case" + to_string(casenum) + ".txt";
        fin.open(case_path);
        if (fin.is_open())
            cout << "open success " << casenum << endl;  // DEBUG
        else
            break;

        int src_id, dst_id;
        for (int i = 0; i < N; i++) {
            getline(fin, str);
            int ptr = 0, j = 0;
            while (j < N) {
                if (str[ptr] == '-') 
                    maze[i * N + j] = BLK, ptr += 3, j++;
                else if (str[ptr] == 'S')
                    maze[i * N + j] = 0, ptr += 2, src_id = i * N + j, j++;
                else if (str[ptr] == 'D')
                    maze[i * N + j] = 0, ptr += 2, dst_id = i * N + j, j++;
                else
                    maze[i * N + j] = (short)(str[ptr] - '0'), ptr += 2, j++;
            }
        }

        for (int i = 0; i < N * N; i++)
            dist[i] = inf, prev[i] = -1, dist_tmp[i] = inf;
        dist[src_id] = 0, prev[src_id] = src_id, dist_tmp[src_id] = 0;


        hipDeviceReset();
        short *d_maze;
        int *d_dist, *d_prev, *d_dist_tmp, *d_dx, *d_dy;
        bool *d_has_next, has_next;
        int dx[4] = {-1, 0, 1, 0};
        int dy[4] = {0, -1, 0, 1};

        // cuda显存的分配，以及把主机上对应的数组内容拷贝到device上
        hipMalloc(&d_maze, sizeof(short) * N * N);
        hipMalloc(&d_dist, sizeof(int) * N * N);
        hipMalloc(&d_prev, sizeof(int) * N * N);
        hipMalloc(&d_dist_tmp, sizeof(int) * N * N);
        hipMalloc(&d_has_next, sizeof(bool));
        hipMalloc(&d_dx, sizeof(int) * 4);
        hipMalloc(&d_dy, sizeof(int) * 4);

        hipMemcpy(d_maze, maze, sizeof(short) * N * N, hipMemcpyHostToDevice);
        hipMemcpy(d_dist, dist, sizeof(int) * N * N, hipMemcpyHostToDevice);
        hipMemcpy(d_prev, prev, sizeof(int) * N * N, hipMemcpyHostToDevice);
        hipMemcpy(d_dist_tmp, dist_tmp, sizeof(int) * N * N, hipMemcpyHostToDevice);
        hipMemcpy(d_dx, dx, sizeof(int) * 4, hipMemcpyHostToDevice);
        hipMemcpy(d_dy, dy, sizeof(int) * 4, hipMemcpyHostToDevice);

        printf("ready to start BF\n");

        timeval startTime, endTime;
        gettimeofday(&startTime, NULL);
        // clock_t startTime = clock();

        for (int i = 0; i < N * N - 1; i++) {  // (N*N-1)轮循环，每轮都是所有线程并行操作的
            has_next = false;
            hipMemcpy(d_has_next, &has_next, sizeof(bool), hipMemcpyHostToDevice);  // 最初，d_has_next是false，松弛时可能会改变
            BF_algo<<<BLOCK_NUM, THREAD_NUM>>>(d_maze, d_dist, d_prev, d_dist_tmp, d_has_next, d_dx, d_dy, N);
            hipDeviceSynchronize();  // 等待所有格子执行结束
            hipMemcpy(&has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);
            if (!has_next) {  // 先判断算法是否结束
                printf("finish in %d loop\n", i);  // DEBUG
                break;
            }
            hipMemcpy(d_dist, d_dist_tmp, sizeof(int) * N * N, hipMemcpyDeviceToDevice);  // 把新的dist_tmp拷贝到dist
            if (i % 2000 == 0) {
                printf("kernel %d\n", i);  // DEBUG
            }
        }

        gettimeofday(&endTime, NULL);
        double consume = (endTime.tv_sec - startTime.tv_sec) + (double)(endTime.tv_usec - startTime.tv_usec) / 1000000.0;
        
        // 结束后，从device上把dist和prev拷贝回主机，准备得出最短路径。然后把cudaMalloc分配的所有资源释放掉
        hipMemcpy(dist, d_dist, sizeof(int) * N * N, hipMemcpyDeviceToHost);
        hipMemcpy(prev, d_prev, sizeof(int) * N * N, hipMemcpyDeviceToHost);
        hipFree(d_maze);
        hipFree(d_dist);
        hipFree(d_prev);
        hipFree(d_dist_tmp);
        hipFree(d_has_next);
        hipFree(d_dx);
        hipFree(d_dy);
 
        printf("cuda end:  dist is %d\n", dist[dst_id]);
        
        if (dist[dst_id] == inf) {
            fout << "case" << casenum << " -1" << endl;
            // consume_time[casenum] = (double)(clock() - startTime) / CLOCKS_PER_SEC;
            continue;
        }

        fout << "case" << casenum << " " << dist[dst_id] << " ";
        vector<int> path;
        path.push_back(dst_id);
        int now_nd = dst_id;
        while(prev[now_nd] != now_nd) {
            path.push_back(prev[now_nd]);
            now_nd = prev[now_nd];
        }

        // clock_t endTime = clock();
        // consume_time[casenum] = (double)(endTime - startTime) / CLOCKS_PER_SEC;
        int sz = path.size();

        for (int i = sz - 1; i >= 0; i--) {
            fout << "(" << path[i] / N << ", " << path[i] % N << ") ";
        }
        fout << endl;
        fin.close();
        cout << "case " << casenum << " " << consume << endl;  // 时间就不单独输出到文件了，直接显示在屏幕上
    }
    
    free(maze);
    free(dist);
    free(prev);
    free(dist_tmp);   

    fout.close();

    return 0;
}
